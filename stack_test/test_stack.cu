#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


//2048(threads)*1024 *8(arrays) *8(sizeof(double)

#define SIZE 1024*4

__global__ void mykernel(double *out, int *in){

  double A0[SIZE];
  double A1[SIZE];

  double res = 0.0;
  int i;
  

  int index = threadIdx.x + blockIdx.x * blockDim.x;

  for(i=0; i<SIZE; i++){

    A0[i] = (double) (index % 25);
    A1[i] = (double) (index % 49);

  }


  for(i=0; i<SIZE; i++){

    A0[in[i]] += A1[i];
    A1[in[i]] += A0[i];

  }

  for(i=0; i<SIZE; i++)
    res += A0[i] + A1[i];


  out[index] = res;

}


int main(){

  int i;
  int nb_threads = 2048;

  double *out = (double *) malloc(nb_threads * sizeof(double));
  int *in     = (int *)    malloc(SIZE * sizeof(int)); 

  
  for(i=0; i<nb_threads; i++)
    out[i] = 0.0;

  for(i=0; i<SIZE; i++)
    in[i] = (i+127) % SIZE;


  double *d_out;
  int    *d_in;

  
  hipMalloc((void **) &d_out, nb_threads * sizeof(double));
  hipMalloc((void **) &d_in , SIZE * sizeof(double));

  hipMemcpy(d_out, out, nb_threads * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_in , in , SIZE * sizeof(int), hipMemcpyHostToDevice);

  dim3 blocksize = 64;
  dim3 numblock  = (nb_threads + blocksize.x -1) / blocksize.x;

  for(i=0; i<1024*10; i++)
    mykernel<<<numblock,blocksize>>>(d_out, d_in);

  hipMemcpy(out, d_out, nb_threads * sizeof(double), hipMemcpyDeviceToHost);

  printf("%lf\n", out[5]);

}
